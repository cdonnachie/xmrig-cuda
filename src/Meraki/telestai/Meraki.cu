#include "hip/hip_runtime.h"
/* XMRig
 * Copyright 2010      Jeff Garzik <jgarzik@pobox.com>
 * Copyright 2012-2014 pooler      <pooler@litecoinpool.org>
 * Copyright 2014      Lucas Jones <https://github.com/lucasjones>
 * Copyright 2014-2016 Wolf9466    <https://github.com/OhGodAPet>
 * Copyright 2016      Jay D Dee   <jayddee246@gmail.com>
 * Copyright 2017-2018 XMR-Stak    <https://github.com/fireice-uk>, <https://github.com/psychocrypt>
 * Copyright 2018-2020 SChernykh   <https://github.com/SChernykh>
 * Copyright 2016-2020 XMRig       <https://github.com/xmrig>, <support@xmrig.com>
 *
 *   This program is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdint>

#include "cryptonight.h"
#include "cuda_device.hpp"
#include "Meraki_dag.h"
#include "CudaMeraki_gen.h"


void meraki_prepare(nvid_ctx *ctx, const void* cache, size_t cache_size, const void* dag_precalc, size_t dag_size, uint32_t height, const uint64_t* dag_sizes)
{
    constexpr size_t MEM_ALIGN = 1024 * 1024;

    if (cache_size != ctx->meraki_cache_size) {
        ctx->meraki_cache_size = cache_size;

        if (!dag_precalc) {
            if (cache_size > ctx->meraki_cache_capacity) {
                CUDA_CHECK(ctx->device_id, hipFree(ctx->meraki_cache));

                ctx->meraki_cache_capacity = ((cache_size + MEM_ALIGN - 1) / MEM_ALIGN) * MEM_ALIGN;
                CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->meraki_cache, ctx->meraki_cache_capacity));
            }

            CUDA_CHECK(ctx->device_id, hipMemcpy((uint8_t*)(ctx->meraki_cache), cache, cache_size, hipMemcpyHostToDevice));
        }
    }

    if (dag_size != ctx->meraki_dag_size) {
        ctx->meraki_dag_size = dag_size;

        if (dag_size > ctx->meraki_dag_capacity) {
            CUDA_CHECK(ctx->device_id, hipFree(ctx->meraki_dag));

            ctx->meraki_dag_capacity = ((dag_size + MEM_ALIGN - 1) / MEM_ALIGN) * MEM_ALIGN;
            CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->meraki_dag, ctx->meraki_dag_capacity));
        }

        if (dag_precalc) {
            CUDA_CHECK(ctx->device_id, hipMemcpy((uint8_t*)(ctx->meraki_dag), cache, cache_size, hipMemcpyHostToDevice));
        }

        constexpr int blocks = 8192;
        constexpr int threads = 32;

        const size_t cache_items = ((cache_size + 255) / 256) * 256 / sizeof(hash64_t);
        const size_t dag_items = dag_size / sizeof(hash64_t);

        uint4 light_words;
        light_words.w = ctx->meraki_cache_size / sizeof(hash64_t);
        Meraki_calculate_fast_mod_data(light_words.w, light_words.x, light_words.y, light_words.z);

        for (size_t i = dag_precalc ? cache_items : 0; i < dag_items; i += blocks * threads) {
            CUDA_CHECK_KERNEL(ctx->device_id, meraki_calculate_dag_item<<<blocks, threads>>>(
                i,
                (hash64_t*) ctx->meraki_dag,
                ctx->meraki_dag_size,
                (hash64_t*)(dag_precalc ? ctx->meraki_dag : ctx->meraki_cache),
                light_words
            ));
            CUDA_CHECK(ctx->device_id, hipDeviceSynchronize());
        }

        if (dag_precalc) {
            CUDA_CHECK(ctx->device_id, hipMemcpy((uint8_t*)(ctx->meraki_dag), dag_precalc, cache_items * sizeof(hash64_t), hipMemcpyHostToDevice));
        }
    }

    constexpr uint32_t PERIOD_LENGTH = 3;
    const uint32_t period = height / PERIOD_LENGTH;

    if (ctx->meraki_period != period) {
        if (ctx->meraki_module) {
            hipModuleUnload(ctx->meraki_module);
        }

        std::vector<char> ptx;
        std::string lowered_name;
        Meraki_get_program(ptx, lowered_name, period, ctx->device_threads, ctx->device_arch[0], ctx->device_arch[1], dag_sizes);

        CU_CHECK(ctx->device_id, hipModuleLoadDataEx(&ctx->meraki_module, ptx.data(), 0, 0, 0));
        CU_CHECK(ctx->device_id, hipModuleGetFunction(&ctx->meraki_kernel, ctx->meraki_module, lowered_name.c_str()));

        ctx->meraki_period = period;

        Meraki_get_program(ptx, lowered_name, period + 1, ctx->device_threads, ctx->device_arch[0], ctx->device_arch[1], dag_sizes, true);
    }

    if (!ctx->meraki_stop_host) {
        CUDA_CHECK(ctx->device_id, hipHostMalloc(&ctx->meraki_stop_host, sizeof(uint32_t) * 2));
        CUDA_CHECK(ctx->device_id, hipHostGetDevicePointer(&ctx->meraki_stop_device, ctx->meraki_stop_host, 0));
    }
}


void meraki_stop_hash(nvid_ctx *ctx)
{
    if (ctx->meraki_stop_host) {
        *ctx->meraki_stop_host = 1;
    }
}


namespace Meraki_Telestai {

void hash(nvid_ctx *ctx, uint8_t* job_blob, uint64_t target, uint32_t *rescount, uint32_t *resnonce, uint32_t *skipped_hashes)
{
    dim3 grid(ctx->device_blocks);
    dim3 block(ctx->device_threads);

    uint32_t hack_false = 0;
    void* args[] = { &ctx->meraki_dag, &ctx->d_input, &target, &hack_false, &ctx->d_result_nonce, &ctx->meraki_stop_device };

    CUDA_CHECK(ctx->device_id, hipMemcpy(ctx->d_input, job_blob, 40, hipMemcpyHostToDevice));
    CUDA_CHECK(ctx->device_id, hipMemset(ctx->d_result_nonce, 0, sizeof(uint32_t)));
    memset(ctx->meraki_stop_host, 0, sizeof(uint32_t) * 2);

    CU_CHECK(ctx->device_id, hipModuleLaunchKernel(
        ctx->meraki_kernel,
        grid.x, grid.y, grid.z,
        block.x, block.y, block.z,
        0, nullptr, args, 0
    ));
    CU_CHECK(ctx->device_id, hipCtxSynchronize());

    *skipped_hashes = ctx->meraki_stop_host[1];

    uint32_t results[16];
    CUDA_CHECK(ctx->device_id, hipMemcpy(results, ctx->d_result_nonce, sizeof(results), hipMemcpyDeviceToHost));

    if (results[0] > 15) {
        results[0] = 15;
    }

    *rescount = results[0];
    memcpy(resnonce, results + 1, results[0] * sizeof(uint32_t));
}

}
